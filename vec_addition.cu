#include <string>                                 //For stoi.
#include <iostream>                               //For stdout.
#include <cstdlib>                                //For random number generator.
#include <chrono>                                 //For getting time.
#include <climits>                                //For maximum n.

#include "hip/hip_runtime.h"                         //For Windows support.


//The type that is used for the calculations.
typedef int type;

//Define constants for min/max.
#define RANDOMIZE_MIN -10
#define RANDOMIZE_MAX 10

//Cuda calculator which will run in each thread.
__global__ void cuda_calculator(type* a, type* b, type* c, int* num_calcs)
{
    //Calculate the starting index.
    size_t start_index = (threadIdx.x + blockIdx.x * blockDim.x) * (*num_calcs);

    //Add the vectors in the current thread index.
    for(size_t i = 0; i < *num_calcs; i++)
        c[start_index + i] = a[start_index + i] + b[start_index + i];
}

//Cuda addition which runs the cuda program.
int cuda_addition(type* a, type* b, type* c, size_t n, size_t blocks,
    size_t threads, double times[3])
{
    //Create pointers for the GPU memory allocation
    type* cu_vec_a;
    type* cu_vec_b;
    type* cu_vec_c;
    int* cu_num_calcs;

    //Calculate the number of elements that this thread will take.
    size_t num_calcs = (n / (blocks * threads));

    //Check if it's not rounded, or it it was zero.
    if(n % (blocks * threads) != 0 || num_calcs <= 0)
        num_calcs++;

    //Allocate memory on the device for the arrays.
    hipMalloc((void**) &cu_vec_a, sizeof(type) * n);
    hipMalloc((void**) &cu_vec_b, sizeof(type) * n);
    hipMalloc((void**) &cu_vec_c, sizeof(type) * n);
    hipMalloc((void**) &cu_num_calcs, sizeof(int));

    //Wait for the thread to finish execution.
    hipDeviceSynchronize();

    //Capture the beginning time before the data transfer (from host).
    auto begin_transfer_to = std::chrono::high_resolution_clock::now();

    //Copy the data, and the size from the main memory to VRAM.
    hipMemcpy(cu_vec_a, a, sizeof(type) * n, hipMemcpyHostToDevice);
    hipMemcpy(cu_vec_b, b, sizeof(type) * n, hipMemcpyHostToDevice);
    hipMemcpy(cu_num_calcs, &num_calcs, sizeof(int), hipMemcpyHostToDevice);

    //Wait for the thread to finish execution.
    hipDeviceSynchronize();

    //Calculate the total time in seconds that it took to transfer data to the device
    auto total_transfer_to = std::chrono::high_resolution_clock::now() - begin_transfer_to;
    times[0] = std::chrono::duration<double> (total_transfer_to).count();

    //Capture the beginning time before the calculations.
    auto begin_calcs_only = std::chrono::high_resolution_clock::now();

    //Launch the addition kernel on the device.
    cuda_calculator<<<blocks, threads>>>(cu_vec_a, cu_vec_b, cu_vec_c, cu_num_calcs);

    //Check if we got any errors.
    if(hipGetLastError() != hipSuccess)
        return EXIT_FAILURE;

    //Wait for the thread to finish execution.
    hipDeviceSynchronize();

    //Calculate the total time in seconds that it took to calculate.
    auto total_calcs_only = std::chrono::high_resolution_clock::now() - begin_calcs_only;
    times[1] = std::chrono::duration<double> (total_calcs_only).count();

    //Capture the beginning time before the calculations.
    auto begin_transfer_from = std::chrono::high_resolution_clock::now();

    //Copy the results back from Vram to main ram.
    hipMemcpy(c, cu_vec_c, sizeof(type) * n, hipMemcpyDeviceToHost);

    //Wait for the thread to finish execution.
    hipDeviceSynchronize();

    //Calculate the total time in seconds that it took to transfer back to host.
    auto total_transfer_from = std::chrono::high_resolution_clock::now() - begin_transfer_from;
    times[2] = std::chrono::duration<double> (total_transfer_from).count();

    //Deallocate memory in the GPU.
    hipFree(cu_vec_a);
    hipFree(cu_vec_b);
    hipFree(cu_vec_c);
    hipFree(cu_num_calcs);

    //Wait for the thread to finish execution.
    hipDeviceSynchronize();

    return EXIT_SUCCESS;
}

//Sequential addition function.
double seq_addition(type* a, type* b, type* c, size_t size)
{
    //Capture the beginning time before the calculations.
    auto begin = std::chrono::high_resolution_clock::now();

    //Iterate over the vectors and add the elements.
    for(size_t i = 0; i < size; i++)
        c[i] = a[i] + b[i];

    //Calculate and return the total time in seconds that it took to compute.
    auto total = std::chrono::high_resolution_clock::now() - begin;
    return std::chrono::duration<double> (total).count();;
}

//Sequential subtraction function (used for residual matrix).
void seq_subtraction(type* a, type* b, type* c, size_t size)
{
    //Iterate over the vectors and subtract the elements.
    for(size_t i = 0; i < size; i++)
        c[i] = a[i] - b[i];
}

//A function which randomizes the vector, by defualt it only uses values between -10 - 10
void randomize(type* vec, int size, int min = RANDOMIZE_MIN, int max = RANDOMIZE_MAX)
{
    //Perform this to ensure the random number generation is truly random.
    std::srand(std::chrono::system_clock::now().time_since_epoch().count());

    //Iterate through, and generate random numbers for each index.
    for(size_t i = 0; i < size; i++)
        vec[i] = ((type) std::rand() %
            (type) (RANDOMIZE_MAX * 2) + (type) RANDOMIZE_MIN) % RANDOMIZE_MAX ;
}

//Print the given vector to stdout.
void dump(type* vec, size_t size)
{
    //Iterate through, and generate random numbers for each index.
    for(size_t i = 0; i < size - 1; i++)
        std::cout << std::scientific << vec[i] <<  " | " ;

    //Print the last item in a different format and add a new line.
    std::cout << std::scientific << vec[size - 1] << std::endl;
}

//A function which will be called when there is an error.
int error(std::string msg)
{
    //Print the error message.
    std::cout << msg << std::endl;

    //Print the usage message.
    std::cout << std::endl << "Usage Guide:" << std::endl
        << "\t* ./a.out <Size of Vectors> <Number of Blocks> <Number of Threads>"
        << " <Output Mode>" << std::endl << "\t* Output mode is either \'q\' "
        << "(quiet) or \'v\' (verbose)" << std::endl
        << "\t* Number of blocks and threads are for the GPU." << std::endl;

    //Return exit failure for passing it back to the terminal.
    return EXIT_FAILURE;
}

//Main method which parses the arguments, and runs the program.
int main(int argc, char** argv)
{
    //Define values for parameters.
    size_t n, blocks, threads;
    bool verbose;

    //Check for invalid number of args.
    if(argc != 5)
        return error("Invalid number of arguments.");

    //Parse the arguments.
    try
    {
        n = std::stoi(argv[1]);
        blocks = std::stoi(argv[2]);
        threads = std::stoi(argv[3]);
    }
    catch(...)      //If we get here, there was an error in the arguments.
    {
        return error("Invalid arguments, could not parse.");
    }

    //Check the print mode.
    if(std::string(argv[4]) == "q" || std::string(argv[4]) == "v")
        //If the mode is valid and set to v, set verbose to true, false otherwise.
        verbose = (std::string(argv[4]) == "v" ? true : false);
    else
        //If we get here an invalid mode was passed.
        return error("Invalid print mode.");

    //Check for invalid threads / blocks / n sizes.
    if(n < 1 || blocks < 1 || threads < 1)
        return error("Invalid arguments. All parameters should be positive.");

    //Check if we're gonna get overflow.
    if(n > UINT_MAX)
        return error("Integer Overflow, please reduce N.");

    //Allocate memory for the input vectors.
    type* vec_a = new type[n];
    type* vec_b = new type[n];

    //Randomize the input vectors.
    randomize(vec_a, n);
    randomize(vec_b, n);

    //Allocate output matrices for the sequential and cuda executions.
    type* vec_c_seq = new type[n];
    type* vec_c_cuda = new type[n];

    //Perform the sequential addition.
    double seq_time = seq_addition(vec_a, vec_b, vec_c_seq, n);

    //Perform the cuda addition, and capture the timings.
    double times[3];
    int stat = cuda_addition(vec_a, vec_b, vec_c_cuda, n, blocks, threads, times);

    //Check the status.
    if(stat == EXIT_FAILURE)
        std::cout << "Error: Failed to execute kernel." << std::endl;

    //Print the timing results, and the input arguments.
    std::cout << "[Cuda_Transfer_To_Device_Seconds]=" << std::scientific << times[0]
        << "  [Cuda_Transfer_To_Host_Seconds]=" << std::scientific << times[2]
        << "  [Cuda_Calculation_Time_Seconds]=" << std::scientific << times[1]
        << "  [Sequential_Time_Seconds]=" << std::scientific << seq_time
        << "  [N]=" << n << "  [Blocks]=" << blocks
        << "  [Threads]=" << threads
        << std::endl;


    //Calculate residual vector for sequential implementation vs cuda.
    type* residual = new type[n];
    seq_subtraction(vec_c_seq, vec_c_cuda, residual, n);

    //Check if we're in verbose output mode.
    if(verbose)
    {
        //Print out the inputs, calculations and residual vector.
        std::cout << std::endl << "Printing out the First Vector:" << std::endl;
        dump(vec_a, n);

        std::cout << "\nPrinting out the Second Vector:" << std::endl;
        dump(vec_b, n);

        std::cout << "\nPrinting out the Addition results (Sequential):" << std::endl;
        dump(vec_c_seq, n);

        std::cout << "\nPrinting out the Addition results (Cuda):" << std::endl;
        dump(vec_c_cuda, n);

        std::cout << "\nPrinting out the residual matrix (Seq - Cuda):" << std::endl;
        dump(residual, n);
    }

    //Deallocate the memory in the heap.
    delete[] vec_a, vec_b, vec_c_seq, vec_c_cuda, residual;

    return EXIT_SUCCESS;
}
